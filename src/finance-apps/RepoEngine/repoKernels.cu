#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// repoKernels.cu
// Scott Grauer-Gray
// Kernels for running Repo on the GPU

#include "repoKernels.cuh"

__device__ int monthLengthKernelGpu(int month, bool leapYear) {
  int MonthLength[12];
  MonthLength[0] = 31;
  MonthLength[1] = 28;
  MonthLength[2] = 31;
  MonthLength[3] = 30;
  MonthLength[4] = 31;
  MonthLength[5] = 30;
  MonthLength[6] = 31;
  MonthLength[7] = 31;
  MonthLength[8] = 30;
  MonthLength[9] = 31;
  MonthLength[10] = 30;
  MonthLength[11] = 31;

  int MonthLeapLength[12];
  MonthLeapLength[0] = 31;
  MonthLeapLength[1] = 29;
  MonthLeapLength[2] = 31;
  MonthLeapLength[3] = 30;
  MonthLeapLength[4] = 31;
  MonthLeapLength[5] = 30;
  MonthLeapLength[6] = 31;
  MonthLeapLength[7] = 31;
  MonthLeapLength[8] = 30;
  MonthLeapLength[9] = 31;
  MonthLeapLength[10] = 30;
  MonthLeapLength[11] = 31;

  return (leapYear ? MonthLeapLength[month - 1] : MonthLength[month - 1]);
}

__device__ int monthOffsetKernelGpu(int m, bool leapYear) {
  int MonthOffset[13];
  MonthOffset[0] = 0;
  MonthOffset[1] = 31;
  MonthOffset[2] = 59;
  MonthOffset[3] = 90;
  MonthOffset[4] = 120;
  MonthOffset[5] = 151;
  MonthOffset[6] = 181;
  MonthOffset[7] = 212;
  MonthOffset[8] = 243;
  MonthOffset[9] = 273;
  MonthOffset[10] = 304;
  MonthOffset[11] = 334;
  MonthOffset[12] = 365;

  int MonthLeapOffset[13];
  MonthLeapOffset[0] = 0;
  MonthLeapOffset[1] = 31;
  MonthLeapOffset[2] = 60;
  MonthLeapOffset[3] = 91;
  MonthLeapOffset[4] = 121;
  MonthLeapOffset[5] = 152;
  MonthLeapOffset[6] = 182;
  MonthLeapOffset[7] = 213;
  MonthLeapOffset[8] = 244;
  MonthLeapOffset[9] = 274;
  MonthLeapOffset[10] = 305;
  MonthLeapOffset[11] = 335;
  MonthLeapOffset[12] = 366;

  return (leapYear ? MonthLeapOffset[m - 1] : MonthOffset[m - 1]);
}

__device__ int yearOffsetKernelGpu(int y) {

  int YearOffset[121];
  YearOffset[0] = 0;
  ;
  YearOffset[1] = 366;
  ;
  YearOffset[2] = 731;
  YearOffset[3] = 1096;
  YearOffset[4] = 1461;
  YearOffset[5] = 1827;
  YearOffset[6] = 2192;
  YearOffset[7] = 2557;
  YearOffset[8] = 2922;
  YearOffset[9] = 3288;
  YearOffset[10] = 3653;
  YearOffset[11] = 4018;
  YearOffset[12] = 4383;
  YearOffset[13] = 4749;
  YearOffset[14] = 5114;
  YearOffset[15] = 5479;
  YearOffset[16] = 5844;
  YearOffset[17] = 6210;
  YearOffset[18] = 6575;
  YearOffset[19] = 6940;
  YearOffset[20] = 7305;
  YearOffset[21] = 7671;

  YearOffset[22] = 8036;
  YearOffset[23] = 8401;
  YearOffset[24] = 8766;
  YearOffset[25] = 9132;
  YearOffset[26] = 9497;
  YearOffset[27] = 9862;
  YearOffset[28] = 10227;
  YearOffset[29] = 10593;
  YearOffset[30] = 10958;
  YearOffset[31] = 11323;
  YearOffset[32] = 11688;
  YearOffset[33] = 12054;

  YearOffset[34] = 12419;
  YearOffset[35] = 12784;
  YearOffset[36] = 13149;
  YearOffset[37] = 13515;
  YearOffset[38] = 13880;
  YearOffset[39] = 14245;
  YearOffset[40] = 14610;
  YearOffset[41] = 14976;
  YearOffset[42] = 15341;
  YearOffset[43] = 15706;
  YearOffset[44] = 16071;
  YearOffset[45] = 16437;

  YearOffset[46] = 16802;
  YearOffset[47] = 17167;
  YearOffset[48] = 17532;
  YearOffset[49] = 17898;
  YearOffset[50] = 18263;
  YearOffset[51] = 18628;
  YearOffset[52] = 18993;
  YearOffset[53] = 19359;
  YearOffset[54] = 19724;
  YearOffset[55] = 20089;
  YearOffset[56] = 20454;
  YearOffset[57] = 20820;

  YearOffset[58] = 21185;
  YearOffset[59] = 21550;
  YearOffset[60] = 21915;
  YearOffset[61] = 22281;
  YearOffset[62] = 22646;
  YearOffset[63] = 23011;
  YearOffset[64] = 23376;
  YearOffset[65] = 23742;
  YearOffset[66] = 24107;
  YearOffset[67] = 24472;
  YearOffset[68] = 24837;
  YearOffset[69] = 25203;

  YearOffset[70] = 25568;
  YearOffset[71] = 25933;
  YearOffset[72] = 26298;
  YearOffset[73] = 26664;
  YearOffset[74] = 27029;
  YearOffset[75] = 27394;
  YearOffset[76] = 27759;
  YearOffset[77] = 28125;
  YearOffset[78] = 28490;
  YearOffset[79] = 28855;
  YearOffset[80] = 29220;
  YearOffset[81] = 29586;

  YearOffset[82] = 29951;
  YearOffset[83] = 30316;
  YearOffset[84] = 30681;
  YearOffset[85] = 31047;
  YearOffset[86] = 31412;
  YearOffset[87] = 31777;
  YearOffset[88] = 32142;
  YearOffset[89] = 32508;
  YearOffset[90] = 32873;
  YearOffset[91] = 33238;
  YearOffset[92] = 33603;
  YearOffset[93] = 33969;

  YearOffset[94] = 34334;
  YearOffset[95] = 34699;
  YearOffset[96] = 35064;
  YearOffset[97] = 35430;
  YearOffset[98] = 35795;
  YearOffset[99] = 36160;
  YearOffset[100] = 36525;
  YearOffset[101] = 36891;
  YearOffset[102] = 37256;
  YearOffset[103] = 37621;
  YearOffset[104] = 37986;
  YearOffset[105] = 38352;

  YearOffset[106] = 38717;
  YearOffset[107] = 39082;
  YearOffset[108] = 39447;
  YearOffset[109] = 39813;
  YearOffset[110] = 40178;
  YearOffset[111] = 40543;
  YearOffset[112] = 40908;
  YearOffset[113] = 41274;
  YearOffset[114] = 41639;
  YearOffset[115] = 42004;
  YearOffset[116] = 42369;
  YearOffset[117] = 42735;
  YearOffset[118] = 43100;
  YearOffset[119] = 42735;
  YearOffset[120] = 43830;

  return YearOffset[y - 1900];
}

__device__ bool isLeapKernelGpu(int y) {
  bool YearIsLeap[121];

  YearIsLeap[0] = 1;
  ;
  YearIsLeap[1] = 0;
  ;
  YearIsLeap[2] = 0;
  YearIsLeap[3] = 0;  // 1096;
  YearIsLeap[4] = 1;  // 1461;
  YearIsLeap[5] = 0;  // 1827;
  YearIsLeap[6] = 0;  // 2192;
  YearIsLeap[7] = 0;  // 2557;
  YearIsLeap[8] = 1;  // 2922;
  YearIsLeap[9] = 0;  // 3288;
  YearIsLeap[10] = 0; // 3653;
  YearIsLeap[11] = 0; // 4018;
  YearIsLeap[12] = 1; // 4383;
  YearIsLeap[13] = 0; // 4749;
  YearIsLeap[14] = 0; // 5114;
  YearIsLeap[15] = 0; // 5479;
  YearIsLeap[16] = 1; // 5844;
  YearIsLeap[17] = 0; // 6210;
  YearIsLeap[18] = 0; // 6575;
  YearIsLeap[19] = 0; // 6940;
  YearIsLeap[20] = 1; // 7305;
  YearIsLeap[21] = 0; // 7671;

  YearIsLeap[22] = 0; // 8036;
  YearIsLeap[23] = 0; // 8401;
  YearIsLeap[24] = 1; // 8766;
  YearIsLeap[25] = 0; // 9132;
  YearIsLeap[26] = 0; // 9497;
  YearIsLeap[27] = 0; // 9862;
  YearIsLeap[28] = 1; // 10227;
  YearIsLeap[29] = 0; // 10593;
  YearIsLeap[30] = 0; // 10958;
  YearIsLeap[31] = 0; // 11323;
  YearIsLeap[32] = 1; // 11688;
  YearIsLeap[33] = 0; // 12054;

  YearIsLeap[34] = 0; // 12419;
  YearIsLeap[35] = 0; // 12784;
  YearIsLeap[36] = 1; // 13149;
  YearIsLeap[37] = 0; // 13515;
  YearIsLeap[38] = 0; // 13880;
  YearIsLeap[39] = 0; // 14245;
  YearIsLeap[40] = 1; // 14610;
  YearIsLeap[41] = 0; // 14976;
  YearIsLeap[42] = 0; // 15341;
  YearIsLeap[43] = 0; // 15706;
  YearIsLeap[44] = 1; // 16071;
  YearIsLeap[45] = 0; // 16437;

  YearIsLeap[46] = 0; // 16802;
  YearIsLeap[47] = 0; // 17167;
  YearIsLeap[48] = 1; // 17532;
  YearIsLeap[49] = 0; // 17898;
  YearIsLeap[50] = 0; // 18263;
  YearIsLeap[51] = 0; // 18628;
  YearIsLeap[52] = 1; // 18993;
  YearIsLeap[53] = 0; // 19359;
  YearIsLeap[54] = 0; // 19724;
  YearIsLeap[55] = 0; // 20089;
  YearIsLeap[56] = 1; // 20454;
  YearIsLeap[57] = 0; // 20820;

  YearIsLeap[58] = 0; // 21185;
  YearIsLeap[59] = 0; // 21550;
  YearIsLeap[60] = 1; // 21915;
  YearIsLeap[61] = 0; // 22281;
  YearIsLeap[62] = 0; // 22646;
  YearIsLeap[63] = 0; // 23011;
  YearIsLeap[64] = 1; // 23376;
  YearIsLeap[65] = 0; // 23742;
  YearIsLeap[66] = 0; // 24107;
  YearIsLeap[67] = 0; // 24472;
  YearIsLeap[68] = 1; // 24837;
  YearIsLeap[69] = 0; // 25203;

  YearIsLeap[70] = 0; // 25568;
  YearIsLeap[71] = 0; // 25933;
  YearIsLeap[72] = 1; // 26298;
  YearIsLeap[73] = 0; // 26664;
  YearIsLeap[74] = 0; // 27029;
  YearIsLeap[75] = 0; // 27394;
  YearIsLeap[76] = 1; // 27759;
  YearIsLeap[77] = 0; // 28125;
  YearIsLeap[78] = 0; // 28490;
  YearIsLeap[79] = 0; // 28855;
  YearIsLeap[80] = 1; // 29220;
  YearIsLeap[81] = 0; // 29586;

  YearIsLeap[82] = 0; // 29951;
  YearIsLeap[83] = 0; // 30316;
  YearIsLeap[84] = 1; // 30681;
  YearIsLeap[85] = 0; // 31047;
  YearIsLeap[86] = 0; // 31412;
  YearIsLeap[87] = 0; // 31777;
  YearIsLeap[88] = 1; // 32142;
  YearIsLeap[89] = 0; // 32508;
  YearIsLeap[90] = 0; // 32873;
  YearIsLeap[91] = 0; // 33238;
  YearIsLeap[92] = 1; // 33603;
  YearIsLeap[93] = 0; // 33969;

  YearIsLeap[94] = 0;  // 34334;
  YearIsLeap[95] = 0;  // 34699;
  YearIsLeap[96] = 1;  // 35064;
  YearIsLeap[97] = 0;  // 35430;
  YearIsLeap[98] = 0;  // 35795;
  YearIsLeap[99] = 0;  // 36160;
  YearIsLeap[100] = 1; // 36525;
  YearIsLeap[101] = 0; // 36891;
  YearIsLeap[102] = 0; // 37256;
  YearIsLeap[103] = 0; // 37621;
  YearIsLeap[104] = 1; // 37986;
  YearIsLeap[105] = 0; // 38352;

  YearIsLeap[106] = 0; // 38717;
  YearIsLeap[107] = 0; // 39082;
  YearIsLeap[108] = 1; // 39447;
  YearIsLeap[109] = 0; // 39813;
  YearIsLeap[110] = 0; // 40178;
  YearIsLeap[111] = 0; // 40543;
  YearIsLeap[112] = 1; // 40908;
  YearIsLeap[113] = 0; // 41274;
  YearIsLeap[114] = 0; // 41639;
  YearIsLeap[115] = 0; // 42004;
  YearIsLeap[116] = 1; // 42369;
  YearIsLeap[117] = 0; // 42735;
  YearIsLeap[118] = 0; // 43100;
  YearIsLeap[119] = 0; // 42735;
  YearIsLeap[120] = 1; // 43830;

  return YearIsLeap[y - 1900];
}

__device__ repoDateStruct intializeDateKernelGpu(int d, int m, int y) {
  repoDateStruct currDate;

  currDate.day = d;
  currDate.month = m;
  currDate.year = y;

  bool leap = isLeapKernelGpu(y);
  int offset = monthOffsetKernelGpu(m, leap);

  currDate.dateSerialNum = d + offset + yearOffsetKernelGpu(y);

  return currDate;
}

__device__ dataType yearFractionGpu(repoDateStruct d1, repoDateStruct d2,
                                    int dayCounter) {
  return dayCountGpu(d1, d2, dayCounter) / 360.0;
}

__device__ int dayCountGpu(repoDateStruct d1, repoDateStruct d2,
                           int dayCounter) {
  if (dayCounter == USE_EXACT_DAY) {
    int dd1 = d1.day, dd2 = d2.day;
    int mm1 = d1.month, mm2 = d2.month;
    int yy1 = d1.year, yy2 = d2.year;

    if (dd2 == 31 && dd1 < 30) {
      dd2 = 1;
      mm2++;
    }

    return 360 * (yy2 - yy1) + 30 * (mm2 - mm1 - 1) + MAX(0, 30 - dd1) +
           MIN(30, dd2);
  }

  else {
    return (d2.dateSerialNum - d1.dateSerialNum);
  }
}

__device__ dataType couponNotionalGpu() { return 100.0; }

__device__ dataType bondNotionalGpu() { return 100.0; }

__device__ dataType fixedRateCouponNominalGpu() { return 100.0; }

__device__ bool eventHasOccurredGpu(repoDateStruct currDate,
                                    repoDateStruct eventDate) {
  if (eventDate.dateSerialNum > currDate.dateSerialNum)
    return true;
  else
    return false;
}

__device__ bool cashFlowHasOccurredGpu(repoDateStruct refDate,
                                       repoDateStruct eventDate) {
  return eventHasOccurredGpu(refDate, eventDate);
}

__device__ repoDateStruct advanceDateGpu(repoDateStruct date,
                                         int numMonthsAdvance) {
  int d = date.day;
  int m = date.month + numMonthsAdvance;
  int y = date.year;

  while (m > 12) {
    m -= 12;
    y += 1;
  }

  while (m < 1) {
    m += 12;
    y -= 1;
  }

  int length = monthLengthKernelGpu(m, isLeapKernelGpu(y));
  if (d > length)
    d = length;

  repoDateStruct newDate = intializeDateKernelGpu(d, m, y);

  return newDate;
}

__device__ int getNumCashFlowsGpu(inArgsStruct inArgs, int repoNum) {
  int numCashFlows = 0;

  // repoDateStruct endDate = inArgs.bond[repoNum].maturityDate;
  repoDateStruct currCashflowDate = inArgs.bond[repoNum].maturityDate;

  while (currCashflowDate.dateSerialNum >
         inArgs.bond[repoNum].startDate.dateSerialNum) {
    numCashFlows++;
    currCashflowDate = advanceDateGpu(currCashflowDate, -6);
  }

  return numCashFlows + 1;
}

__device__ dataType getDirtyPriceGpu(inArgsStruct inArgs, int repoNum,
                                     cashFlowsStruct cashFlows, int numLegs) {
  dataType currentNotional = bondNotionalGpu();
  return discountingBondEngineCalculateSettlementValueGpu(inArgs, repoNum,
                                                          cashFlows, numLegs) *
         100.0 / currentNotional;
}

__device__ dataType getAccruedAmountGpu(inArgsStruct inArgs,
                                        repoDateStruct date, int repoNum,
                                        cashFlowsStruct cashFlows,
                                        int numLegs) {
  return bondAccruedAmountGpu(inArgs, date, repoNum, cashFlows, numLegs);
}

__device__ dataType discountingBondEngineCalculateSettlementValueGpu(
    inArgsStruct inArgs, int repoNum, cashFlowsStruct cashFlows, int numLegs) {
  repoDateStruct settlementDate = inArgs.settlementDate[repoNum];

  if (settlementDate.dateSerialNum <
      inArgs.bond[repoNum].startDate.dateSerialNum) {
    settlementDate = inArgs.bond[repoNum].startDate;
  }

  return cashFlowsNpvGpu(cashFlows, inArgs.discountCurve[repoNum], false,
                         settlementDate, settlementDate, numLegs);
}

__device__ dataType bondAccruedAmountGpu(inArgsStruct inArgs,
                                         repoDateStruct date, int repoNum,
                                         cashFlowsStruct cashFlows,
                                         int numLegs) {
  dataType currentNotional = bondNotionalGpu();
  if (currentNotional == 0.0)
    return 0.0;

  return bondFunctionsAccruedAmountGpu(inArgs, date, repoNum, cashFlows,
                                       numLegs);
}

__device__ dataType bondFunctionsAccruedAmountGpu(inArgsStruct inArgs,
                                                  repoDateStruct date,
                                                  int repoNum,
                                                  cashFlowsStruct cashFlows,
                                                  int numLegs) {

  return cashFlowsAccruedAmountGpu(cashFlows, false, date, numLegs, inArgs,
                                   repoNum) *
         100.0 / bondNotionalGpu();
}

__device__ dataType cashFlowsAccruedAmountGpu(cashFlowsStruct cashFlows,
                                              bool includeSettlementDateFlows,
                                              repoDateStruct settlementDate,
                                              int numLegs, inArgsStruct inArgs,
                                              int repoNum) {
  int legComputeNum =
      cashFlowsNextCashFlowNumGpu(cashFlows, settlementDate, numLegs);

  dataType result = 0.0;

  int i;
  for (i = legComputeNum; i < (numLegs); ++i) {

    result += fixedRateCouponAccruedAmountGpu(cashFlows, i, settlementDate,
                                              inArgs, repoNum);
  }

  return result;
}

__device__ dataType fixedRateCouponAccruedAmountGpu(cashFlowsStruct cashFlows,
                                                    int numLeg,
                                                    repoDateStruct d,
                                                    inArgsStruct inArgs,
                                                    int repoNum) {
  if (d.dateSerialNum <=
          cashFlows.legs[numLeg].accrualStartDate.dateSerialNum ||
      d.dateSerialNum > inArgs.maturityDate[repoNum].dateSerialNum) {
    return 0.0;
  } else {
    repoDateStruct endDate = cashFlows.legs[numLeg].accrualEndDate;
    if (d.dateSerialNum < cashFlows.legs[numLeg].accrualEndDate.dateSerialNum) {
      endDate = d;
    }

    return fixedRateCouponNominalGpu() *
           (interestRateCompoundFactorFourArgsGpu(
                cashFlows.intRate, cashFlows.legs[numLeg].accrualStartDate,
                endDate, cashFlows.dayCounter) -
            1.0);
  }
}

__device__ dataType cashFlowsNpvGpu(cashFlowsStruct cashFlows,
                                    repoYieldTermStruct discountCurve,
                                    bool includeSettlementDateFlows,
                                    repoDateStruct settlementDate,
                                    repoDateStruct npvDate, int numLegs) {
  npvDate = settlementDate;

  dataType totalNPV = 0.0;

  int i;
  for (i = 0; i < numLegs; ++i) {

    if (!(cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate,
                                 settlementDate)))
      totalNPV += fixedRateCouponAmountGpu(cashFlows, i) *
                  repoYieldTermStructureDiscountGpu(
                      discountCurve, cashFlows.legs[i].paymentDate);
  }

  return totalNPV / repoYieldTermStructureDiscountGpu(discountCurve, npvDate);
}

__device__ dataType repoYieldTermStructureDiscountGpu(
    repoYieldTermStruct ytStruct, repoDateStruct t) {
  ytStruct.intRate.rate = ytStruct.forward;
  ytStruct.intRate.freq = ytStruct.frequency;
  ytStruct.intRate.comp = ytStruct.compounding;
  return flatForwardDiscountImplGpu(
      ytStruct.intRate,
      yearFractionGpu(ytStruct.refDate, t, ytStruct.dayCounter));
}

__device__ dataType flatForwardDiscountImplGpu(intRateStruct intRate,
                                               dataType t) {
  return interestRateDiscountFactorGpu(intRate, t);
}

__device__ dataType interestRateDiscountFactorGpu(intRateStruct intRate,
                                                  dataType t) {
  return 1.0 / interestRateCompoundFactorGpu(intRate, t);
}

__device__ dataType interestRateCompoundFactorGpu(intRateStruct intRate,
                                                  dataType t) {
  /// switch (intRate.comp)
  {
    if (intRate.comp == SIMPLE_INTEREST)
      return 1.0 + intRate.rate * t;
    else if (intRate.comp == COMPOUNDED_INTEREST)
      return pow(1.0f + intRate.rate / intRate.freq, intRate.freq * t);
    else if (intRate.comp == CONTINUOUS_INTEREST)
      return exp(intRate.rate * t);
    // case SimpleThenCompounded:
    //  if (t<=1.0/Real(freq_))
    //      return 1.0 + intRate.rate*t;
    //  else
    //      return pow(1.0+r_/freq_, freq_*t);
  }

  return 0.0f;
}

__device__ dataType fixedRateCouponAmountGpu(cashFlowsStruct cashFlows,
                                             int numLeg) {
  if (cashFlows.legs[numLeg].amount == COMPUTE_AMOUNT) {
    return fixedRateCouponNominalGpu() *
           (interestRateCompoundFactorFourArgsGpu(
                cashFlows.intRate, cashFlows.legs[numLeg].accrualStartDate,
                cashFlows.legs[numLeg].accrualEndDate, cashFlows.dayCounter) -
            1.0);
  } else {
    return cashFlows.legs[numLeg].amount;
  }
}

__device__ dataType interestRateCompoundFactorFourArgsGpu(intRateStruct intRate,
                                                          repoDateStruct d1,
                                                          repoDateStruct d2,
                                                          int dayCounter) {
  dataType t = yearFractionGpu(d1, d2, dayCounter);
  return interestRateCompoundFactorGpu(intRate, t);
}

__device__ dataType fixedRateBondForwardSpotIncomeGpu(inArgsStruct inArgs,
                                                      int repoNum,
                                                      cashFlowsStruct cashFlows,
                                                      int numLegs) {
  dataType income = 0.0;
  repoDateStruct settlement = inArgs.settlementDate[repoNum];
  repoDateStruct maturityDate = inArgs.deliveryDate[repoNum];
  repoYieldTermStruct repoCurve = inArgs.repoCurve[repoNum];

  int i;
  for (i = 0; i < numLegs; ++i) {

    if (!cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate, settlement)) {
      if (cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate, maturityDate)) {
        income += fixedRateCouponAmountGpu(cashFlows, i) *
                  repoYieldTermStructureDiscountGpu(
                      repoCurve, cashFlows.legs[i].paymentDate);
      } else {
        break;
      }
    }
  }

  return income;
}

__device__ dataType getImpliedYieldGpu(inArgsStruct inArgs,
                                       dataType forwardValue,
                                       dataType underlyingSpotValue,
                                       dataType spotIncomeIncDiscCurve,
                                       int repoNum) {
  dataType t = yearFractionGpu(inArgs.settlementDate[repoNum],
                               inArgs.repoDeliveryDate[repoNum],
                               inArgs.repoCurve[repoNum].dayCounter);
  dataType compoundingFactor =
      forwardValue / (underlyingSpotValue - spotIncomeIncDiscCurve);
  return interestRateImpliedRateGpu(compoundingFactor, SIMPLE_INTEREST, 1.0, t);
}

__device__ dataType interestRateImpliedRateGpu(dataType compound, int comp,
                                               dataType freq, dataType t) {
  dataType r = 0.0f;
  if (compound == 1.0) {

    r = 0.0;
  } else {
    if (comp == SIMPLE_INTEREST)
      r = (compound - 1.0) / t;
    else if (comp == COMPOUNDED_INTEREST)
      r = (pow((dataType)compound, 1.0f / ((freq)*t)) - 1.0f) * (freq);
  }

  return r;
}

__device__ dataType getMarketRepoRateGpu(repoDateStruct d, int comp,
                                         dataType freq,
                                         repoDateStruct referenceDate,
                                         inArgsStruct inArgs, int repoNum) {
  dataType compound =
      1.0 / repoYieldTermStructureDiscountGpu(inArgs.repoCurve[repoNum], d);
  return interestRateImpliedRateGpu(
      compound, comp, freq,
      yearFractionGpu(referenceDate, d, inArgs.repoCurve[repoNum].dayCounter));
}

__device__ couponStruct cashFlowsNextCashFlowGpu(cashFlowsStruct cashFlows,
                                                 repoDateStruct settlementDate,
                                                 int numLegs) {
  int i;
  for (i = 0; i < numLegs; ++i) {

    if (!(cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate,
                                 settlementDate)))
      return cashFlows.legs[i];
  }
  return cashFlows.legs[numLegs - 1];
}

__device__ int cashFlowsNextCashFlowNumGpu(cashFlowsStruct cashFlows,
                                           repoDateStruct settlementDate,
                                           int numLegs) {
  // int numNext = -1;

  int ii = 0;
  while ((cashFlows.legs[ii].paymentDate.dateSerialNum <
          settlementDate.dateSerialNum) &&
         (ii < (numLegs - 1))) {
    ii = ii + 1;
  }

  return ii;
}

__device__ dataType getBondYieldGpu(dataType cleanPrice, int dc, int comp,
                                    dataType freq, repoDateStruct settlement,
                                    dataType accuracy, int maxEvaluations,
                                    inArgsStruct currInArgs, int repoNum,
                                    cashFlowsStruct cashFlows, int numLegs) {
  dataType currentNotional = bondNotionalGpu();

  if (currentNotional == 0.0)
    return 0.0;

  if (currInArgs.bond[repoNum].startDate.dateSerialNum >
      settlement.dateSerialNum) {
    settlement = currInArgs.bond[repoNum].startDate;
  }

  return getBondFunctionsYieldGpu(cleanPrice, dc, comp, freq, settlement,
                                  accuracy, maxEvaluations, currInArgs, repoNum,
                                  cashFlows, numLegs);
}

__device__ dataType
getBondFunctionsYieldGpu(dataType cleanPrice, int dc, int comp, dataType freq,
                         repoDateStruct settlement, dataType accuracy,
                         int maxEvaluations, inArgsStruct currInArgs,
                         int repoNum, cashFlowsStruct cashFlows, int numLegs) {
  dataType dirtyPrice =
      cleanPrice + bondFunctionsAccruedAmountGpu(currInArgs, settlement,
                                                 repoNum, cashFlows, numLegs);
  dirtyPrice /= 100.0 / bondNotionalGpu();

  return getCashFlowsYieldGpu(cashFlows, dirtyPrice, dc, comp, freq, false,
                              settlement, settlement, numLegs, accuracy,
                              maxEvaluations, 0.05f);
}

__device__ dataType getCashFlowsYieldGpu(
    cashFlowsStruct leg, dataType npv, int dayCounter, int compounding,
    dataType frequency, bool includeSettlementDateFlows,
    repoDateStruct settlementDate, repoDateStruct npvDate, int numLegs,
    dataType accuracy, int maxIterations, dataType guess) {
  // Brent solver;
  solverStruct solver;
  solver.maxEvaluations_ = maxIterations;
  irrFinderStruct objFunction;

  objFunction.npv = npv;
  objFunction.dayCounter = dayCounter;
  objFunction.comp = compounding;
  objFunction.freq = frequency;
  objFunction.includeSettlementDateFlows = includeSettlementDateFlows;
  objFunction.settlementDate = settlementDate;
  objFunction.npvDate = npvDate;

  return solverSolveGpu(solver, objFunction, accuracy, guess, guess / 10.0, leg,
                        numLegs);
}

__device__ dataType solverSolveGpu(solverStruct solver, irrFinderStruct f,
                                   dataType accuracy, dataType guess,
                                   dataType step, cashFlowsStruct cashFlows,
                                   int numLegs) {
  // check whether we really want to use epsilon
  accuracy = MAX(accuracy, QL_EPSILON_GPU);

  dataType growthFactor = 1.6;
  int flipflop = -1;

  solver.root_ = guess;
  solver.fxMax_ = fOpGpu(f, solver.root_, cashFlows, numLegs);

  // monotonically crescent bias, as in optionValue(volatility)
  if (closeGpu(solver.fxMax_, 0.0)) {
    return solver.root_;
  } else if (closeGpu(solver.fxMax_, 0.0)) {
    solver.xMin_ = /*enforceBounds*/ (solver.root_ - step);
    solver.fxMin_ = fOpGpu(f, solver.xMin_, cashFlows, numLegs);
    solver.xMax_ = solver.root_;
  } else {
    solver.xMin_ = solver.root_;
    solver.fxMin_ = solver.fxMax_;
    solver.xMax_ = /*enforceBounds*/ (solver.root_ + step);
    solver.fxMax_ = fOpGpu(f, solver.xMax_, cashFlows, numLegs);
  }

  solver.evaluationNumber_ = 2;
  while (solver.evaluationNumber_ <= solver.maxEvaluations_) {
    if (solver.fxMin_ * solver.fxMax_ <= 0.0) {
      if (closeGpu(solver.fxMin_, 0.0))
        return solver.xMin_;
      if (closeGpu(solver.fxMax_, 0.0))
        return solver.xMax_;
      solver.root_ = (solver.xMax_ + solver.xMin_) / 2.0;
      return solveImplGpu(solver, f, accuracy, cashFlows, numLegs);
    }
    if (fabs(solver.fxMin_) < fabs(solver.fxMax_)) {
      solver.xMin_ = /*enforceBounds*/ (
          solver.xMin_ + growthFactor * (solver.xMin_ - solver.xMax_));
      solver.fxMin_ = fOpGpu(f, solver.xMin_, cashFlows, numLegs);
    } else if (fabs(solver.fxMin_) > fabs(solver.fxMax_)) {
      solver.xMax_ = /*enforceBounds*/ (
          solver.xMax_ + growthFactor * (solver.xMax_ - solver.xMin_));
      solver.fxMax_ = fOpGpu(f, solver.xMax_, cashFlows, numLegs);
    } else if (flipflop == -1) {
      solver.xMin_ = /*enforceBounds*/ (
          solver.xMin_ + growthFactor * (solver.xMin_ - solver.xMax_));
      solver.fxMin_ = fOpGpu(f, solver.xMin_, cashFlows, numLegs);
      solver.evaluationNumber_++;
      flipflop = 1;
    } else if (flipflop == 1) {
      solver.xMax_ = /*enforceBounds*/ (
          solver.xMax_ + growthFactor * (solver.xMax_ - solver.xMin_));
      solver.fxMax_ = fOpGpu(f, solver.xMax_, cashFlows, numLegs);
      flipflop = -1;
    }
    solver.evaluationNumber_++;
  }

  return 0.0f;
}

__device__ dataType cashFlowsNpvYieldGpu(cashFlowsStruct cashFlows,
                                         intRateStruct y,
                                         bool includeSettlementDateFlows,
                                         repoDateStruct settlementDate,
                                         repoDateStruct npvDate, int numLegs) {

  dataType npv = 0.0;
  dataType discount = 1.0;
  repoDateStruct lastDate;
  bool first = true;

  int i;

  for (i = 0; i < numLegs; ++i) {

    if (cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate, settlementDate))
      continue;

    repoDateStruct couponDate = cashFlows.legs[i].paymentDate;
    dataType amount = fixedRateCouponAmountGpu(cashFlows, i);
    if (first) {
      first = false;

      if (i > 0) {
        lastDate = advanceDateGpu(cashFlows.legs[i].paymentDate, -1 * 6);
      } else {
        lastDate = cashFlows.legs[i].accrualStartDate;
      }

      discount *= interestRateDiscountFactorGpu(
          y, yearFractionGpu(npvDate, couponDate, y.dayCounter));

    } else {
      discount *= interestRateDiscountFactorGpu(
          y, yearFractionGpu(lastDate, couponDate, y.dayCounter));
    }

    lastDate = couponDate;

    npv += amount * discount;
  }

  return npv;
}

__device__ dataType fOpGpu(irrFinderStruct f, dataType y,
                           cashFlowsStruct cashFlows, int numLegs) {
  intRateStruct yield;

  yield.rate = y;
  yield.comp = f.comp;
  yield.freq = f.freq;
  yield.dayCounter = f.dayCounter;

  dataType NPV = cashFlowsNpvYieldGpu(cashFlows, yield, false, f.settlementDate,
                                      f.npvDate, numLegs);

  return (f.npv - NPV);
}

__device__ dataType fDerivativeGpu(irrFinderStruct f, dataType y,
                                   cashFlowsStruct cashFlows, int numLegs) {
  intRateStruct yield;
  yield.rate = y;
  yield.dayCounter = f.dayCounter;
  yield.comp = f.comp;
  yield.freq = f.freq;

  return modifiedDurationGpu(cashFlows, yield, f.includeSettlementDateFlows,
                             f.settlementDate, f.npvDate, numLegs);
}

__device__ bool closeGpu(dataType x, dataType y) {
  return closeThreeArgsGpu(x, y, 42);
}

__device__ bool closeThreeArgsGpu(dataType x, dataType y, int n) {
  dataType diff = fabs(x - y);
  dataType tolerance = n * QL_EPSILON_GPU;

  return diff <= tolerance * fabs(x) && diff <= tolerance * fabs(y);
}

__device__ dataType enforceBoundsGpu(dataType x) {
  /*if (lowerBoundEnforced_ && x < lowerBound_)
          return lowerBound_;
  if (upperBoundEnforced_ && x > upperBound_)
          return upperBound_;*/
  return x;
}

__device__ dataType solveImplGpu(solverStruct solver, irrFinderStruct f,
                                 dataType xAccuracy, cashFlowsStruct cashFlows,
                                 int numLegs) {
  dataType froot, dfroot, dx, dxold;
  dataType xh, xl;

  // Orient the search so that f(xl) < 0
  if (solver.fxMin_ < 0.0) {
    xl = solver.xMin_;
    xh = solver.xMax_;
  } else {
    xh = solver.xMin_;
    xl = solver.xMax_;
  }

  // the "stepsize before last"
  dxold = solver.xMax_ - solver.xMin_;
  // it was dxold=std::fabs(xMax_-xMin_); in Numerical Recipes
  // here (xMax_-xMin_ > 0) is verified in the constructor

  // and the last step
  dx = dxold;

  froot = fOpGpu(f, solver.root_, cashFlows, numLegs);
  dfroot = fDerivativeGpu(f, solver.root_, cashFlows, numLegs);

  ++solver.evaluationNumber_;

  while (solver.evaluationNumber_ <= solver.maxEvaluations_) {
    // Bisect if (out of range || not decreasing fast enough)
    if ((((solver.root_ - xh) * dfroot - froot) *
             ((solver.root_ - xl) * dfroot - froot) >
         0.0) ||
        (fabs(2.0 * froot) > fabs(dxold * dfroot))) {
      dxold = dx;
      dx = (xh - xl) / 2.0;
      solver.root_ = xl + dx;
    } else {
      dxold = dx;
      dx = froot / dfroot;
      solver.root_ -= dx;
    }

    // Convergence criterion
    if (fabs(dx) < xAccuracy)
      return solver.root_;
    froot = fOpGpu(f, solver.root_, cashFlows, numLegs);
    dfroot = fDerivativeGpu(f, solver.root_, cashFlows, numLegs);
    ++solver.evaluationNumber_;
    if (froot < 0.0)
      xl = solver.root_;
    else
      xh = solver.root_;
  }

  return solver.root_;
}

__device__ dataType modifiedDurationGpu(cashFlowsStruct cashFlows,
                                        intRateStruct y,
                                        bool includeSettlementDateFlows,
                                        repoDateStruct settlementDate,
                                        repoDateStruct npvDate, int numLegs) {
  dataType P = 0.0;
  dataType dPdy = 0.0;
  dataType r = y.rate;
  dataType N = y.freq;
  int dc = y.dayCounter;

  int i;
  for (i = 0; i < numLegs; ++i) {

    if (!cashFlowHasOccurredGpu(cashFlows.legs[i].paymentDate,
                                settlementDate)) {
      dataType t = yearFractionGpu(npvDate, cashFlows.legs[i].paymentDate, dc);
      dataType c = fixedRateCouponAmountGpu(cashFlows, i);
      dataType B = interestRateDiscountFactorGpu(y, t);

      P += c * B;
      // switch (y.comp)
      {
        if (y.comp == SIMPLE_INTEREST)
          dPdy -= c * B * B * t;
        //	break;
        else if (y.comp == COMPOUNDED_INTEREST)
          dPdy -= c * t * B / (1 + r / N);
        //	break;
        else if (y.comp == CONTINUOUS_INTEREST)
          dPdy -= c * B * t;
        //	break;
        else if (y.comp == SIMPLE_THEN_COMPOUNDED_INTEREST) {
          if (t <= 1.0 / N)
            dPdy -= c * B * B * t;
          else
            dPdy -= c * t * B / (1 + r / N);
        }
        //	break;
      }
    }
  }

  if (P == 0.0) // no cashflows
  {
    return 0.0;
  }
  return (-1 * dPdy) / P; // reverse derivative sign
}

__global__ void getRepoResultsGpu(inArgsStruct inArgs, resultsStruct results,
                                  int n) {
  int repoNum = blockIdx.x * blockDim.x + threadIdx.x;

  if (repoNum < n) {
    const int numLegs = getNumCashFlowsGpu(inArgs, repoNum);
    cashFlowsStruct cashFlows;
    couponStruct legs[9]; // originally numLegs

    cashFlows.legs = legs;
    cashFlows.intRate.dayCounter = USE_EXACT_DAY;
    cashFlows.intRate.rate = inArgs.bond[repoNum].rate;
    cashFlows.intRate.freq = ANNUAL_FREQ;
    cashFlows.intRate.comp = SIMPLE_INTEREST;
    cashFlows.dayCounter = USE_EXACT_DAY;
    cashFlows.nominal = 100.0;

    // repoDateStruct currPaymentDate;
    repoDateStruct currStartDate =
        advanceDateGpu(inArgs.bond[repoNum].maturityDate, (numLegs - 1) * -6);
    repoDateStruct currEndDate = advanceDateGpu(currStartDate, 6);
    int cashFlowNum;

    for (cashFlowNum = 0; cashFlowNum < numLegs - 1; cashFlowNum++) {
      cashFlows.legs[cashFlowNum].paymentDate = currEndDate;
      cashFlows.legs[cashFlowNum].accrualStartDate = currStartDate;
      cashFlows.legs[cashFlowNum].accrualEndDate = currEndDate;
      cashFlows.legs[cashFlowNum].amount = COMPUTE_AMOUNT;
      currStartDate = currEndDate;
      currEndDate = advanceDateGpu(currEndDate, 6);
    }

    cashFlows.legs[numLegs - 1].paymentDate = inArgs.bond[repoNum].maturityDate;
    cashFlows.legs[numLegs - 1].accrualStartDate =
        inArgs.settlementDate[repoNum];
    cashFlows.legs[numLegs - 1].accrualEndDate = inArgs.settlementDate[repoNum];
    cashFlows.legs[numLegs - 1].amount = 100.0;

    results.bondForwardVal[repoNum] = getBondYieldGpu(
        inArgs.bondCleanPrice[repoNum], USE_EXACT_DAY, COMPOUNDED_INTEREST, 2.0,
        inArgs.settlementDate[repoNum], ACCURACY, 100, inArgs, repoNum,
        cashFlows, numLegs);

    inArgs.discountCurve[repoNum].forward = results.bondForwardVal[repoNum];

    results.dirtyPrice[repoNum] =
        getDirtyPriceGpu(inArgs, repoNum, cashFlows, numLegs);

    results.accruedAmountSettlement[repoNum] = getAccruedAmountGpu(
        inArgs, inArgs.settlementDate[repoNum], repoNum, cashFlows, numLegs);

    results.accruedAmountDeliveryDate[repoNum] = getAccruedAmountGpu(
        inArgs, inArgs.deliveryDate[repoNum], repoNum, cashFlows, numLegs);

    results.cleanPrice[repoNum] =
        results.dirtyPrice[repoNum] - results.accruedAmountSettlement[repoNum];

    results.forwardSpotIncome[repoNum] =
        fixedRateBondForwardSpotIncomeGpu(inArgs, repoNum, cashFlows, numLegs);

    results.underlyingBondFwd[repoNum] =
        results.forwardSpotIncome[repoNum] /
        repoYieldTermStructureDiscountGpu(inArgs.repoCurve[repoNum],
                                          inArgs.repoDeliveryDate[repoNum]);

    dataType forwardVal =
        (results.dirtyPrice[repoNum] - results.forwardSpotIncome[repoNum]) /
        repoYieldTermStructureDiscountGpu(inArgs.repoCurve[repoNum],
                                          inArgs.repoDeliveryDate[repoNum]);

    results.repoNpv[repoNum] =
        (forwardVal - inArgs.dummyStrike[repoNum]) *
        repoYieldTermStructureDiscountGpu(inArgs.repoCurve[repoNum],
                                          inArgs.repoDeliveryDate[repoNum]);

    results.repoCleanForwardPrice[repoNum] =
        forwardVal - getAccruedAmountGpu(inArgs,
                                         inArgs.repoDeliveryDate[repoNum],
                                         repoNum, cashFlows, numLegs);

    results.repoDirtyForwardPrice[repoNum] = forwardVal;
    results.repoImpliedYield[repoNum] = getImpliedYieldGpu(
        inArgs, inArgs.dummyStrike[repoNum], results.dirtyPrice[repoNum],
        results.forwardSpotIncome[repoNum], repoNum);
    results.marketRepoRate[repoNum] = getMarketRepoRateGpu(
        inArgs.repoDeliveryDate[repoNum], SIMPLE_INTEREST, 1.0,
        inArgs.settlementDate[repoNum], inArgs, repoNum);
  }
}
