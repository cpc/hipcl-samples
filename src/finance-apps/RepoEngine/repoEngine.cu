#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
// repoEngine.cu
// Scott Grauer-Gray
// Engine for running the repo (repurchase agreement) on the GPU

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <stdio.h>
#include <sys/time.h>

#include "repoKernels.cuh"
#include "repoKernelsCpu.cuh"
#include "repoStructs.cuh"

#include <time.h>

#define MIN(a, b) (((a) < (b)) ? (a) : (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))

//#define NUM_REPOS_RUN 1000000
#define NUM_REPOS_RUN 1000

int monthLengthCpu(int month, bool leapYear) {
  int MonthLength[] = {31, 28, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31};

  int MonthLeapLength[] = {31, 29, 31, 30, 31, 30, 31, 31, 30, 31, 30, 31};

  return (leapYear ? MonthLeapLength[month - 1] : MonthLength[month - 1]);
}

int monthOffsetCpu(int m, bool leapYear) {
  int MonthOffset[] = {
      0,   31,  59,  90,  120, 151, // Jan - Jun
      181, 212, 243, 273, 304, 334, // Jun - Dec
      365                           // used in dayOfMonth to bracket day
  };

  int MonthLeapOffset[] = {
      0,   31,  60,  91,  121, 152, // Jan - Jun
      182, 213, 244, 274, 305, 335, // Jun - Dec
      366                           // used in dayOfMonth to bracket day
  };

  return (leapYear ? MonthLeapOffset[m - 1] : MonthOffset[m - 1]);
}

int yearOffsetCpu(int y) {
  // the list of all December 31st in the preceding year
  // e.g. for 1901 yearOffset[1] is 366, that is, December 31 1900
  int YearOffset[] = {
      // 1900-1909
      0, 366, 731, 1096, 1461, 1827, 2192, 2557, 2922, 3288,
      // 1910-1919
      3653, 4018, 4383, 4749, 5114, 5479, 5844, 6210, 6575, 6940,
      // 1920-1929
      7305, 7671, 8036, 8401, 8766, 9132, 9497, 9862, 10227, 10593,
      // 1930-1939
      10958, 11323, 11688, 12054, 12419, 12784, 13149, 13515, 13880, 14245,
      // 1940-1949
      14610, 14976, 15341, 15706, 16071, 16437, 16802, 17167, 17532, 17898,
      // 1950-1959
      18263, 18628, 18993, 19359, 19724, 20089, 20454, 20820, 21185, 21550,
      // 1960-1969
      21915, 22281, 22646, 23011, 23376, 23742, 24107, 24472, 24837, 25203,
      // 1970-1979
      25568, 25933, 26298, 26664, 27029, 27394, 27759, 28125, 28490, 28855,
      // 1980-1989
      29220, 29586, 29951, 30316, 30681, 31047, 31412, 31777, 32142, 32508,
      // 1990-1999
      32873, 33238, 33603, 33969, 34334, 34699, 35064, 35430, 35795, 36160,
      // 2000-2009
      36525, 36891, 37256, 37621, 37986, 38352, 38717, 39082, 39447, 39813,
      // 2010-2019
      40178, 40543, 40908, 41274, 41639, 42004, 42369, 42735, 43100, 43465,
      // 2020-2029
      43830, 44196, 44561, 44926, 45291, 45657, 46022, 46387, 46752, 47118,
      // 2030-2039
      47483, 47848, 48213, 48579, 48944, 49309, 49674, 50040, 50405, 50770,
      // 2040-2049
      51135, 51501, 51866, 52231, 52596, 52962, 53327, 53692, 54057, 54423,
      // 2050-2059
      54788, 55153, 55518, 55884, 56249, 56614, 56979, 57345, 57710, 58075,
      // 2060-2069
      58440, 58806, 59171, 59536, 59901, 60267, 60632, 60997, 61362, 61728,
      // 2070-2079
      62093, 62458, 62823, 63189, 63554, 63919, 64284, 64650, 65015, 65380,
      // 2080-2089
      65745, 66111, 66476, 66841, 67206, 67572, 67937, 68302, 68667, 69033,
      // 2090-2099
      69398, 69763, 70128, 70494, 70859, 71224, 71589, 71955, 72320, 72685,
      // 2100-2109
      73050, 73415, 73780, 74145, 74510, 74876, 75241, 75606, 75971, 76337,
      // 2110-2119
      76702, 77067, 77432, 77798, 78163, 78528, 78893, 79259, 79624, 79989,
      // 2120-2129
      80354, 80720, 81085, 81450, 81815, 82181, 82546, 82911, 83276, 83642,
      // 2130-2139
      84007, 84372, 84737, 85103, 85468, 85833, 86198, 86564, 86929, 87294,
      // 2140-2149
      87659, 88025, 88390, 88755, 89120, 89486, 89851, 90216, 90581, 90947,
      // 2150-2159
      91312, 91677, 92042, 92408, 92773, 93138, 93503, 93869, 94234, 94599,
      // 2160-2169
      94964, 95330, 95695, 96060, 96425, 96791, 97156, 97521, 97886, 98252,
      // 2170-2179
      98617, 98982, 99347, 99713, 100078, 100443, 100808, 101174, 101539,
      101904,
      // 2180-2189
      102269, 102635, 103000, 103365, 103730, 104096, 104461, 104826, 105191,
      105557,
      // 2190-2199
      105922, 106287, 106652, 107018, 107383, 107748, 108113, 108479, 108844,
      109209,
      // 2200
      109574};

  return YearOffset[y - 1900];
}

bool isLeapCpu(int y) {
  bool YearIsLeap[] = {
      // 1900 is leap in agreement with Excel's bug
      // 1900 is out of valid date range anyway
      // 1900-1909
      true, false, false, false, true, false, false, false, true, false,
      // 1910-1919
      false, false, true, false, false, false, true, false, false, false,
      // 1920-1929
      true, false, false, false, true, false, false, false, true, false,
      // 1930-1939
      false, false, true, false, false, false, true, false, false, false,
      // 1940-1949
      true, false, false, false, true, false, false, false, true, false,
      // 1950-1959
      false, false, true, false, false, false, true, false, false, false,
      // 1960-1969
      true, false, false, false, true, false, false, false, true, false,
      // 1970-1979
      false, false, true, false, false, false, true, false, false, false,
      // 1980-1989
      true, false, false, false, true, false, false, false, true, false,
      // 1990-1999
      false, false, true, false, false, false, true, false, false, false,
      // 2000-2009
      true, false, false, false, true, false, false, false, true, false,
      // 2010-2019
      false, false, true, false, false, false, true, false, false, false,
      // 2020-2029
      true, false, false, false, true, false, false, false, true, false,
      // 2030-2039
      false, false, true, false, false, false, true, false, false, false,
      // 2040-2049
      true, false, false, false, true, false, false, false, true, false,
      // 2050-2059
      false, false, true, false, false, false, true, false, false, false,
      // 2060-2069
      true, false, false, false, true, false, false, false, true, false,
      // 2070-2079
      false, false, true, false, false, false, true, false, false, false,
      // 2080-2089
      true, false, false, false, true, false, false, false, true, false,
      // 2090-2099
      false, false, true, false, false, false, true, false, false, false,
      // 2100-2109
      false, false, false, false, true, false, false, false, true, false,
      // 2110-2119
      false, false, true, false, false, false, true, false, false, false,
      // 2120-2129
      true, false, false, false, true, false, false, false, true, false,
      // 2130-2139
      false, false, true, false, false, false, true, false, false, false,
      // 2140-2149
      true, false, false, false, true, false, false, false, true, false,
      // 2150-2159
      false, false, true, false, false, false, true, false, false, false,
      // 2160-2169
      true, false, false, false, true, false, false, false, true, false,
      // 2170-2179
      false, false, true, false, false, false, true, false, false, false,
      // 2180-2189
      true, false, false, false, true, false, false, false, true, false,
      // 2190-2199
      false, false, true, false, false, false, true, false, false, false,
      // 2200
      false};

  return YearIsLeap[y - 1900];
}

repoDateStruct intializeDateCpu(int d, int m, int y) {
  repoDateStruct currDate;

  currDate.day = d;
  currDate.month = m;
  currDate.year = y;

  bool leap = isLeapCpu(y);
  int offset = monthOffsetCpu(m, leap);

  currDate.dateSerialNum = d + offset + yearOffsetCpu(y);

  return currDate;
}

void runRepoEngine() {
  // can run multiple times with different number of bonds by uncommenting these
  // lines int nReposArray[] = {100, 200, 500, 1000, 2000, 5000, 10000, 20000,
  // 50000, 100000, 200000, 500000, 1000000, 2000000};

  // for (int numTime=0; numTime < 14; numTime++)
  {

    int numRepos = NUM_REPOS_RUN; // nReposArray[numTime];
    printf("\nNUM_REPOS: %d\n\n", numRepos);

    inArgsStruct inArgsHost;

    inArgsHost.discountCurve =
        (repoYieldTermStruct *)malloc(numRepos * sizeof(repoYieldTermStruct));
    inArgsHost.repoCurve =
        (repoYieldTermStruct *)malloc(numRepos * sizeof(repoYieldTermStruct));
    inArgsHost.settlementDate =
        (repoDateStruct *)malloc(numRepos * sizeof(repoDateStruct));
    inArgsHost.deliveryDate =
        (repoDateStruct *)malloc(numRepos * sizeof(repoDateStruct));
    inArgsHost.maturityDate =
        (repoDateStruct *)malloc(numRepos * sizeof(repoDateStruct));
    inArgsHost.repoDeliveryDate =
        (repoDateStruct *)malloc(numRepos * sizeof(repoDateStruct));
    inArgsHost.bondCleanPrice = (dataType *)malloc(numRepos * sizeof(dataType));
    inArgsHost.bond = (bondStruct *)malloc(numRepos * sizeof(bondStruct));
    inArgsHost.dummyStrike = (dataType *)malloc(numRepos * sizeof(dataType));

    srand(time(NULL));

    for (int numRepo = 0; numRepo < numRepos; numRepo++) {
      repoDateStruct repoSettlementDate =
          intializeDateCpu(rand() % 28 + 1, 3 - (rand() % 3), 2000);
      repoDateStruct repoDeliveryDate =
          intializeDateCpu(rand() % 28 + 1, 9 + (rand() % 3), 2000);
      dataType repoRate = 0.05 + ((float)rand() / (float)RAND_MAX - 0.5) * 0.1;

      // int repoSettlementDays = 0;
      int repoCompounding = SIMPLE_INTEREST;
      dataType repoCompoundFreq = 1;

      repoDateStruct bondIssueDate = intializeDateCpu(
          rand() % 28 + 1, rand() % 12 + 1, 1999 - (rand() % 2));
      // repoDateStruct bondDatedDate = bondIssueDate;
      repoDateStruct bondMaturityDate =
          intializeDateCpu(rand() % 28 + 1, rand() % 12 + 1, 2001);

      bondStruct bond;
      bond.startDate = bondIssueDate;
      bond.maturityDate = bondMaturityDate;
      bond.rate = 0.08 + ((float)rand() / (float)RAND_MAX - 0.5) * 0.1;

      // dataType bondCoupon = bond.rate;
      dataType bondCouponFrequency = 2;

      // int bondSettlementDays = 0;

      dataType bondCleanPrice = 89.97693786;
      // dataType bondRedemption = 100.0;
      // dataType faceAmount = 100.0;

      repoYieldTermStruct bondCurve;

      bondCurve.refDate = repoSettlementDate;
      bondCurve.calDate = repoSettlementDate;
      bondCurve.forward = -0.1f; // dummy rate
      bondCurve.compounding = COMPOUNDED_INTEREST;
      bondCurve.frequency = bondCouponFrequency;
      bondCurve.dayCounter = USE_EXACT_DAY;
      bondCurve.refDate = repoSettlementDate;
      bondCurve.calDate = repoSettlementDate;
      bondCurve.compounding = COMPOUNDED_INTEREST;
      bondCurve.frequency = bondCouponFrequency;

      dataType dummyStrike = 91.5745;

      repoYieldTermStruct repoCurve;
      repoCurve.refDate = repoSettlementDate;
      repoCurve.calDate = repoSettlementDate;
      repoCurve.forward = repoRate;
      repoCurve.compounding = repoCompounding;
      repoCurve.frequency = repoCompoundFreq;
      repoCurve.dayCounter = USE_SERIAL_NUMS;

      inArgsHost.discountCurve[numRepo] = bondCurve;
      inArgsHost.repoCurve[numRepo] = repoCurve;
      inArgsHost.settlementDate[numRepo] = repoSettlementDate;
      inArgsHost.deliveryDate[numRepo] = repoDeliveryDate;
      inArgsHost.maturityDate[numRepo] = bondMaturityDate;
      inArgsHost.repoDeliveryDate[numRepo] = repoDeliveryDate;
      inArgsHost.bondCleanPrice[numRepo] = bondCleanPrice;
      inArgsHost.bond[numRepo] = bond;
      inArgsHost.dummyStrike[numRepo] = dummyStrike;
    }

    printf("Inputs for repo with index %d\n", numRepos / 2);
    printf("Repo Settlement Date: %d-%d-%d\n",
           inArgsHost.settlementDate[numRepos / 2].month,
           inArgsHost.settlementDate[numRepos / 2].day,
           inArgsHost.settlementDate[numRepos / 2].year);
    printf("Repo Delivery Date: %d-%d-%d\n",
           inArgsHost.deliveryDate[numRepos / 2].month,
           inArgsHost.deliveryDate[numRepos / 2].day,
           inArgsHost.deliveryDate[numRepos / 2].year);
    printf("Bond Issue Date: %d-%d-%d\n",
           inArgsHost.bond[numRepos / 2].startDate.month,
           inArgsHost.bond[numRepos / 2].startDate.day,
           inArgsHost.bond[numRepos / 2].startDate.year);
    printf("Bond Maturity Date: %d-%d-%d\n",
           inArgsHost.maturityDate[numRepos / 2].month,
           inArgsHost.maturityDate[numRepos / 2].day,
           inArgsHost.maturityDate[numRepos / 2].year);
    printf("Repo rate: %f\n", inArgsHost.repoCurve[numRepos / 2].forward);
    printf("Bond rate: %f\n", inArgsHost.bond[numRepos / 2].rate);
    printf("Bond clean price: %f\n\n", inArgsHost.bondCleanPrice[numRepos / 2]);

    resultsStruct resultsHost;
    resultsStruct resultsFromGpu;

    resultsHost.dirtyPrice = (dataType *)malloc(numRepos * sizeof(dataType));
    resultsHost.accruedAmountSettlement =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.accruedAmountDeliveryDate =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.cleanPrice = (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.forwardSpotIncome =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.underlyingBondFwd =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.repoNpv = (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.repoCleanForwardPrice =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.repoDirtyForwardPrice =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.repoImpliedYield =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.marketRepoRate =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsHost.bondForwardVal =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;

    resultsFromGpu.dirtyPrice = (dataType *)malloc(numRepos * sizeof(dataType));
    resultsFromGpu.accruedAmountSettlement =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.accruedAmountDeliveryDate =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.cleanPrice = (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.forwardSpotIncome =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.underlyingBondFwd =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.repoNpv = (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.repoCleanForwardPrice =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.repoDirtyForwardPrice =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.repoImpliedYield =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.marketRepoRate =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;
    resultsFromGpu.bondForwardVal =
        (dataType *)malloc(numRepos * sizeof(dataType));
    ;

    inArgsStruct inArgsGpu;
    resultsStruct resultsGpu;

    hipMalloc((void **)&(resultsGpu.dirtyPrice), numRepos * sizeof(dataType));
    hipMalloc((void **)&(resultsGpu.accruedAmountSettlement),
              numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.accruedAmountDeliveryDate),
              numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.cleanPrice), numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.forwardSpotIncome),
              numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.underlyingBondFwd),
              numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.repoNpv), numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.repoCleanForwardPrice),
              numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.repoDirtyForwardPrice),
              numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.repoImpliedYield),
              numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.marketRepoRate),
              numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(resultsGpu.bondForwardVal),
              numRepos * sizeof(dataType));
    ;

    hipMalloc((void **)&(inArgsGpu.discountCurve),
              numRepos * sizeof(repoYieldTermStruct));
    ;
    hipMalloc((void **)&(inArgsGpu.repoCurve),
              numRepos * sizeof(repoYieldTermStruct));
    ;
    hipMalloc((void **)&(inArgsGpu.settlementDate),
              numRepos * sizeof(repoDateStruct));
    ;
    hipMalloc((void **)&(inArgsGpu.deliveryDate),
              numRepos * sizeof(repoDateStruct));
    ;
    hipMalloc((void **)&(inArgsGpu.maturityDate),
              numRepos * sizeof(repoDateStruct));
    ;
    hipMalloc((void **)&(inArgsGpu.repoDeliveryDate),
              numRepos * sizeof(repoDateStruct));
    ;
    hipMalloc((void **)&(inArgsGpu.bondCleanPrice),
              numRepos * sizeof(dataType));
    ;
    hipMalloc((void **)&(inArgsGpu.bond), numRepos * sizeof(bondStruct));
    ;
    hipMalloc((void **)&(inArgsGpu.dummyStrike), numRepos * sizeof(dataType));
    ;

    dim3 blockDim(32, 1);
    dim3 gridDim((size_t)ceil((dataType)numRepos / (dataType)blockDim.x), 1);

    hipMemcpy((inArgsGpu.discountCurve), inArgsHost.discountCurve,
              numRepos * sizeof(repoYieldTermStruct), hipMemcpyHostToDevice);
    ;
    hipMemcpy((inArgsGpu.repoCurve), inArgsHost.repoCurve,
              numRepos * sizeof(repoYieldTermStruct), hipMemcpyHostToDevice);
    ;
    hipMemcpy((inArgsGpu.settlementDate), inArgsHost.settlementDate,
              numRepos * sizeof(repoDateStruct), hipMemcpyHostToDevice);
    ;
    hipMemcpy((inArgsGpu.deliveryDate), inArgsHost.deliveryDate,
              numRepos * sizeof(repoDateStruct), hipMemcpyHostToDevice);
    ;
    hipMemcpy((inArgsGpu.maturityDate), inArgsHost.maturityDate,
              numRepos * sizeof(repoDateStruct), hipMemcpyHostToDevice);
    ;
    hipMemcpy((inArgsGpu.repoDeliveryDate), inArgsHost.repoDeliveryDate,
              numRepos * sizeof(repoDateStruct), hipMemcpyHostToDevice);
    ;
    hipMemcpy((inArgsGpu.bondCleanPrice), inArgsHost.bondCleanPrice,
              numRepos * sizeof(dataType), hipMemcpyHostToDevice);
    ;
    hipMemcpy((inArgsGpu.bond), inArgsHost.bond, numRepos * sizeof(bondStruct),
              hipMemcpyHostToDevice);
    ;
    hipMemcpy((inArgsGpu.dummyStrike), inArgsHost.dummyStrike,
              numRepos * sizeof(dataType), hipMemcpyHostToDevice);
    ;

    long seconds, useconds;
    float mtimeGpu;
    float mtimeCpu;

    struct timeval start;
    gettimeofday(&start, NULL);

    hipLaunchKernelGGL(getRepoResultsGpu, dim3(gridDim), dim3(blockDim), 0, 0,
                       inArgsGpu, resultsGpu, numRepos);

    hipDeviceSynchronize();
    struct timeval end;
    gettimeofday(&end, NULL);

    seconds = end.tv_sec - start.tv_sec;
    useconds = end.tv_usec - start.tv_usec;

    mtimeGpu = ((seconds)*1000 + ((float)useconds) / 1000.0) + 0.5f;

    printf("Run on GPU\n");
    printf("Processing time on GPU: %f (ms)  \n\n", mtimeGpu);

    // copy data back to CPU
    hipMemcpy(resultsFromGpu.dirtyPrice, (resultsGpu.dirtyPrice),
              numRepos * sizeof(dataType), hipMemcpyDeviceToHost);
    hipMemcpy(resultsFromGpu.accruedAmountSettlement,
              (resultsGpu.accruedAmountSettlement), numRepos * sizeof(dataType),
              hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.accruedAmountDeliveryDate,
              (resultsGpu.accruedAmountDeliveryDate),
              numRepos * sizeof(dataType), hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.cleanPrice, (resultsGpu.cleanPrice),
              numRepos * sizeof(dataType), hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.forwardSpotIncome, (resultsGpu.forwardSpotIncome),
              numRepos * sizeof(dataType), hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.underlyingBondFwd, (resultsGpu.underlyingBondFwd),
              numRepos * sizeof(dataType), hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.repoNpv, (resultsGpu.repoNpv),
              numRepos * sizeof(dataType), hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.repoCleanForwardPrice,
              (resultsGpu.repoCleanForwardPrice), numRepos * sizeof(dataType),
              hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.repoDirtyForwardPrice,
              (resultsGpu.repoDirtyForwardPrice), numRepos * sizeof(dataType),
              hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.repoImpliedYield, (resultsGpu.repoImpliedYield),
              numRepos * sizeof(dataType), hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.marketRepoRate, (resultsGpu.marketRepoRate),
              numRepos * sizeof(dataType), hipMemcpyDeviceToHost);
    ;
    hipMemcpy(resultsFromGpu.bondForwardVal, (resultsGpu.bondForwardVal),
              numRepos * sizeof(dataType), hipMemcpyDeviceToHost);
    ;
    hipDeviceSynchronize();

    double totPrice = 0.0;
    for (int numRepo = 0; numRepo < numRepos; numRepo++) {
      totPrice += resultsFromGpu.dirtyPrice[numRepo];
    }
    printf("Sum of repo dirty price on GPU: %f\n\n", totPrice);
    printf("Computed info on GPU for Repo at index %d\n", numRepos / 2);
    printf("Dirty Price: %f\n", resultsFromGpu.dirtyPrice[numRepos / 2]);
    printf("Accrued Amount: %f\n",
           resultsFromGpu.accruedAmountSettlement[numRepos / 2]);
    printf("Accrued Amount at delivery: %f\n",
           resultsFromGpu.accruedAmountDeliveryDate[numRepos / 2]);
    printf("Clean Price: %f\n", resultsFromGpu.cleanPrice[numRepos / 2]);
    printf("Forward Spot Income: %f\n",
           resultsFromGpu.forwardSpotIncome[numRepos / 2]);
    printf("Underlying Forward Income: %f\n",
           resultsFromGpu.underlyingBondFwd[numRepos / 2]);
    printf("Repo NPV: %f\n", resultsFromGpu.repoNpv[numRepos / 2]);
    printf("Repo Clean Forward Price: %f\n",
           resultsFromGpu.repoCleanForwardPrice[numRepos / 2]);
    printf("Repo Dirty Forward Price: %f\n",
           resultsFromGpu.repoDirtyForwardPrice[numRepos / 2]);
    printf("Repo Implied Yield: %f\n",
           resultsFromGpu.repoImpliedYield[numRepos / 2]);
    printf("Market Repo Rate: %f\n",
           resultsFromGpu.marketRepoRate[numRepos / 2]);
    printf("Bond Forward Val: %f\n\n",
           resultsFromGpu.bondForwardVal[numRepos / 2]);

    gettimeofday(&start, NULL);
    getRepoResultsGpuCpu(inArgsHost, resultsHost, numRepos);
    gettimeofday(&end, NULL);

    totPrice = 0.0;
    for (int numRepo = 0; numRepo < numRepos; numRepo++) {
      totPrice += resultsHost.dirtyPrice[numRepo];
    }

    seconds = end.tv_sec - start.tv_sec;
    useconds = end.tv_usec - start.tv_usec;

    mtimeCpu = ((seconds)*1000 + ((float)useconds) / 1000.0) + 0.5f;

    printf("Run on CPU\n");
    printf("Processing time on CPU: %f (ms)  \n\n", mtimeCpu);

    printf("Sum of repo dirty price on CPU: %f\n\n", totPrice);
    printf("Computed info on CPU for Repo at index %d\n", numRepos / 2);
    printf("Dirty Price: %f\n", resultsHost.dirtyPrice[numRepos / 2]);
    printf("Accrued Amount: %f\n",
           resultsHost.accruedAmountSettlement[numRepos / 2]);
    printf("Accrued Amount at delivery: %f\n",
           resultsHost.accruedAmountDeliveryDate[numRepos / 2]);
    printf("Clean Price: %f\n", resultsHost.cleanPrice[numRepos / 2]);
    printf("Forward Spot Income: %f\n",
           resultsHost.forwardSpotIncome[numRepos / 2]);
    printf("Underlying Forward Income: %f\n",
           resultsHost.underlyingBondFwd[numRepos / 2]);
    printf("Repo NPV: %f\n", resultsHost.repoNpv[numRepos / 2]);
    printf("Repo Clean Forward Price: %f\n",
           resultsHost.repoCleanForwardPrice[numRepos / 2]);
    printf("Repo Dirty Forward Price: %f\n",
           resultsHost.repoDirtyForwardPrice[numRepos / 2]);
    printf("Repo Implied Yield: %f\n",
           resultsHost.repoImpliedYield[numRepos / 2]);
    printf("Market Repo Rate: %f\n", resultsHost.marketRepoRate[numRepos / 2]);
    printf("Bond Forward Val: %f\n\n",
           resultsHost.bondForwardVal[numRepos / 2]);

    printf("Speedup using GPU: %f\n", mtimeCpu / mtimeGpu);

    hipFree(resultsGpu.dirtyPrice);
    hipFree(resultsGpu.accruedAmountSettlement);
    ;
    hipFree(resultsGpu.accruedAmountDeliveryDate);
    ;
    hipFree(resultsGpu.cleanPrice);
    ;
    hipFree(resultsGpu.forwardSpotIncome);
    ;
    hipFree(resultsGpu.underlyingBondFwd);
    ;
    hipFree(resultsGpu.repoNpv);
    ;
    hipFree(resultsGpu.repoCleanForwardPrice);
    ;
    hipFree(resultsGpu.repoDirtyForwardPrice);
    ;
    hipFree(resultsGpu.repoImpliedYield);
    ;
    hipFree(resultsGpu.marketRepoRate);
    ;
    hipFree(resultsGpu.bondForwardVal);
    ;

    hipFree(inArgsGpu.discountCurve);
    hipFree(inArgsGpu.repoCurve);
    hipFree(inArgsGpu.settlementDate);
    hipFree(inArgsGpu.deliveryDate);
    hipFree(inArgsGpu.maturityDate);
    hipFree(inArgsGpu.repoDeliveryDate);
    hipFree(inArgsGpu.bondCleanPrice);
    ;
    hipFree(inArgsGpu.bond);
    hipFree(inArgsGpu.dummyStrike);
    ;

    free(resultsHost.dirtyPrice);
    free(resultsHost.accruedAmountSettlement);
    ;
    free(resultsHost.accruedAmountDeliveryDate);
    ;
    free(resultsHost.cleanPrice);
    ;
    free(resultsHost.forwardSpotIncome);
    ;
    free(resultsHost.underlyingBondFwd);
    ;
    free(resultsHost.repoNpv);
    ;
    free(resultsHost.repoCleanForwardPrice);
    ;
    free(resultsHost.repoDirtyForwardPrice);
    ;
    free(resultsHost.repoImpliedYield);
    ;
    free(resultsHost.marketRepoRate);
    ;
    free(resultsHost.bondForwardVal);
    ;

    free(resultsFromGpu.dirtyPrice);
    free(resultsFromGpu.accruedAmountSettlement);
    ;
    free(resultsFromGpu.accruedAmountDeliveryDate);
    ;
    free(resultsFromGpu.cleanPrice);
    ;
    free(resultsFromGpu.forwardSpotIncome);
    ;
    free(resultsFromGpu.underlyingBondFwd);
    ;
    free(resultsFromGpu.repoNpv);
    ;
    free(resultsFromGpu.repoCleanForwardPrice);
    ;
    free(resultsFromGpu.repoDirtyForwardPrice);
    ;
    free(resultsFromGpu.repoImpliedYield);
    ;
    free(resultsFromGpu.marketRepoRate);
    ;
    free(resultsFromGpu.bondForwardVal);

    free(inArgsHost.discountCurve);
    free(inArgsHost.repoCurve);
    free(inArgsHost.settlementDate);
    free(inArgsHost.deliveryDate);
    free(inArgsHost.maturityDate);
    free(inArgsHost.repoDeliveryDate);
    free(inArgsHost.bondCleanPrice);
    free(inArgsHost.bond);
    free(inArgsHost.dummyStrike);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  runRepoEngine();
  return 0;
}
